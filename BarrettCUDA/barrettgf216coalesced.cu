#include "hip/hip_runtime.h"
// This file is part of BarretCUDA v0.1 
// 
// BarretCUDA is a fast(ish) CUDA implementation of sparse matrix
// multiplication modulo a multi-precision prime.
// 
// Copyright (C) 2016, Ryan Henry and Syed Mahbub Hafiz
// 
// 
// BarretCUDA is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published
// by the Free Software Foundation, either version 3 of the License,
// or (at your option) any later version.
// 
// BarretCUDA is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with BarretCUDA.  If not, see <http://www.gnu.org/licenses/>.
#include <iostream>
#include <hip/hip_runtime.h>
#include <atomic>
#include <chrono>
#include <ratio>
#include <fstream>
#include "gf2earrays.h"
#include "barret.h"

using namespace std;

#define THREADS_PER_BLOCK(n) 	(n >= 512 ? 512 : n)
#define NUM_BLOCKS(n)	((n + THREADS_PER_BLOCK(n) - 1) / THREADS_PER_BLOCK(n))

#define DEBUG_IDX 4

typedef uint16_t uintX;

int u;

__device__ __forceinline__ uint16_t multiply_GF216_Element(const uint16_t x, const uint16_t y,  const uint16_t * d_GF216_log_table, const uint16_t * d_GF216_exp_table)
{
    if(x == 0 || y == 0) return 0;
    return d_GF216_exp_table[d_GF216_log_table[x]+d_GF216_log_table[y]];
}

 void Check_CUDA_Error(const char *message)
{
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess) {
       fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
       exit(-1);
    }                         
}

void SpMV_gf216_cpu(int u_wrap, uint16_t * response, const uint16_t * query, const SparseMatrix<uint> & matrix)
{
	for (int i = 0; i < matrix.ncols; i++)
	{
	     uint col = matrix.l_cols[i];
	     uint16_t temp = 0;
		
	    for (int j = 0; j < u_wrap; j++)
	    {   
		const uint val = matrix.l_vals[i + j * matrix.ncols];
		//if(i==101) printf("CPU: val: %u, col: %u, rows: %u, query: %u, temp: %u\n",(val      )& 0xFFFF,col,matrix.l_rows[col],query[matrix.l_rows[col]],multiply_GF2E<uint16_t>(((val      )& 0xFFFF), query[matrix.l_rows[col]]));
		temp ^= multiply_GF2E<uint16_t>(((val      )& 0xFFFF), query[matrix.l_rows[col++]]);
		//if(i==101) printf("CPU: val: %u, col: %u, rows: %u, query: %u, temp: %u\n",(val >> 16)& 0xFFFF,col,matrix.l_rows[col],query[matrix.l_rows[col]],multiply_GF2E<uint16_t>(((val  >> 16)& 0xFFFF), query[matrix.l_rows[col]]));	
		temp ^= multiply_GF2E<uint16_t>(((val >> 16)& 0xFFFF), query[matrix.l_rows[col++]]);
			
	    }
	    response[i] = temp;
	if(i==101)		printf("CPU response[%d]: %u\n",i,response[i]);
	}
}

__global__ void SpMV_kernel(int u_wrap, uint16_t * response, const uint16_t * query, const uint nvals,
	const uint * vals, const uint ncols, const uint * cols, const uint * rows,  const uint16_t * d_GF216_log_table, const uint16_t * d_GF216_exp_table)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= ncols) return;
    register uint col = cols[i];
    register uint16_t temp = 0;
    	for (int j = 0; j < u_wrap; j++)
    	{
		register const uint val = vals[i + j * ncols];
		//if(i==101) printf("Kernel: val: %u, col: %u, rows: %u, query: %u, temp: %u\n",(val      )& 0xFFFF,col,rows[col],query[rows[col]],multiply_GF216_Element(val      & 0xFFFF, query[rows[col]],d_GF216_log_table,d_GF216_exp_table));
		temp ^= multiply_GF216_Element(((val      )& 0xFFFF), query[rows[col++]],d_GF216_log_table,d_GF216_exp_table);
		//if(i==101) printf("Kernel: val: %u, col: %u, rows: %u, query: %u, temp: %u\n",(val >> 16)& 0xFFFF,col,rows[col],query[rows[col]],multiply_GF216_Element(((val  >> 16)& 0xFFFF), query[rows[col]],d_GF216_log_table,d_GF216_exp_table));	
		temp ^= multiply_GF216_Element(((val >> 16)& 0xFFFF), query[rows[col++]],d_GF216_log_table,d_GF216_exp_table);
	}
    response[i] = temp;
	if(i == 101) printf("GPU response[%d]: %u\n",i,response[i]);
}

void SpMV(int u_wrap, uint16_t * l_response, const uint16_t * l_query,
	uint16_t * d_response, uint16_t * d_query, const hipStream_t & stream,
	const SparseMatrix<uint> & matrix, const uint16_t * d_GF216_log_table, const uint16_t * d_GF216_exp_table)
{
    hipMemcpyAsync(d_query, l_query, matrix.nrows*sizeof(uint16_t), hipMemcpyHostToDevice, stream);
    Check_CUDA_Error("GF216:hipMemcpyAsync:hipMemcpyHostToDevice");
    const dim3 Dg(NUM_BLOCKS(matrix.ncols), 1, 1);
    const dim3 Db(THREADS_PER_BLOCK(matrix.ncols), 1, 1);
    const size_t Ns = 0;

    SpMV_kernel<<< Dg, Db, Ns, stream >>> (u_wrap, d_response, d_query, matrix.nvals, matrix.d_vals, matrix.ncols, matrix.d_cols, matrix.d_rows, d_GF216_log_table,d_GF216_exp_table);
    Check_CUDA_Error("GF216:SpMV_kernel");
    hipMemcpyAsync(l_response, d_response, matrix.ncols*sizeof(uint16_t), hipMemcpyDeviceToHost, stream);
    Check_CUDA_Error("GF216:hipMemcpyAsync:hipMemcpyDeviceToHost");
}
template <typename T>
void initMatrix(const char * valfile, const char * rowfile,
	const char * colfile, NTL::ZZ & modulus,
	struct SparseMatrix<T> & matrix)
{

    std::ifstream rowstream(rowfile, std::ifstream::in);
    if (!rowstream) { cerr << "Error: opening ROWS files\n"; exit(-1); }
    std::ifstream colstream(colfile, std::ifstream::in);
    if (!colstream) { cerr << "Error: opening COLS files\n"; exit(-1); }
    std::ifstream valstream(valfile, std::ifstream::in);
    if (!valstream) { cerr << "Error: opening VALS files\n"; exit(-1); }

    	NTL::ZZ temp_zz;
	valstream >> temp_zz;
	modulus = NTL::trunc_ZZ(temp_zz, 16);

    //valstream >> u;
    rowstream >> matrix.nrows;
    rowstream >> matrix.nvals;
    colstream >> matrix.ncols;
    matrix.l_cols = (uint *)malloc((matrix.ncols+1) * sizeof(uint));
    hipMalloc((void**)&matrix.d_cols, (matrix.ncols+1) * sizeof(uint));

    int u_wrap = ((u-1)/2+1);
  //std::cout << "nvals: " << matrix.nvals << " Size l_vals: "<< u_wrap*matrix.ncols* sizeof(uint) <<" u_wrap really: "<< u_wrap <<"\n";
    matrix.l_rows = (uint *)malloc(matrix.ncols * u * sizeof(uint));memset(matrix.l_rows+matrix.nvals,0,(matrix.ncols * u-matrix.nvals) * sizeof(uint));
    hipMalloc((void**)&matrix.d_rows, matrix.ncols * u * sizeof(uint));
    matrix.l_vals = (uint *)malloc(u_wrap*matrix.ncols* sizeof(uint)); memset(matrix.l_vals,0,u_wrap*matrix.ncols* sizeof(uint));
    hipMalloc((void**)&matrix.d_vals, u_wrap*matrix.ncols* sizeof(uint));



//std::cout << "modulus:\t" << modulus << "\n";
//std::cout << "matrix.nrows:\t" << matrix.nrows << "\n";
//std::cout << "matrix.ncols:\t" << matrix.ncols << "\n";
//std::cout << "matrix.nvals:\t" << matrix.nvals << "\n";

    for (int i = 0; i < matrix.ncols+1; i++)
    {
	colstream >> matrix.l_cols[i];
    }
    colstream.close();


    hipMemcpy(matrix.d_cols, matrix.l_cols, (matrix.ncols+1) * sizeof(uint),
	hipMemcpyHostToDevice);



    NTL::ZZ_pPush p(modulus);
    for (int i = 0; i < matrix.nvals; i++)
    {
	rowstream >> matrix.l_rows[i];
    }
    rowstream.close();
    /*for (int i = matrix.nvals; i < matrix.ncols * u; i++ ) 
    {
	matrix.l_rows[i] = 0;
    }*/
    hipMemcpy(matrix.d_rows, matrix.l_rows, matrix.nvals* sizeof(uint),
	hipMemcpyHostToDevice);

	for(int i=0,c=0;i<matrix.ncols;i++)
	{	int number_of_nnz_in_a_col = matrix.l_cols[i+1] - matrix.l_cols[i];
		//std::cout<< "col i: " << number_of_nnz_in_a_col;
		for(int j=0; j < number_of_nnz_in_a_col;j++)
		{	c++;
			
			NTL::ZZ tmp_zz;
			valstream >> tmp_zz;
			uint tmp = NTL::trunc_long(tmp_zz, 16);
			matrix.l_vals[i+(j/2)*matrix.ncols] +=  tmp<<(16*(j%2));		
//			std::cout << "\nc: " << c << " at ("<< i+(j/2)*matrix.ncols << ", " << 8*(j%2) << ")";
		}
		//std::cout << "\n";

	}
	//to_uint<T>(NTL::rep(tmp), matrix.l_vals[i]);
    valstream.close();
  //std::cout << "problem9\n\n";
    hipMemcpy(matrix.d_vals, matrix.l_vals, u_wrap*matrix.ncols* sizeof(uint),
	hipMemcpyHostToDevice);
//  std::cout << "problem10\n\n";


}

template <typename T>
void freeMatrix(struct SparseMatrix<T> & matrix)
{
    free(matrix.l_vals);
    free(matrix.l_rows);
    free(matrix.l_cols);
    hipFree(matrix.d_vals);
    hipFree(matrix.d_cols);
    hipFree(matrix.d_rows);
}
int main(int argc, char ** argv)
{
    int nstreams = 2;

    u = atoi(argv[4]);

    int u_wrap = ((u-1)/2+1);
    if (argc < 3)
    {
	std::cout << "Usage: " << argv[0] << " VALUES ROWS COLS\n\n";
	return 1;
    }

    time_t t0 = time(0);
    NTL::SetSeed(to_ZZ(t0));
    std::cout << "Seed: " << t0 << "\n";

    struct SparseMatrix<uint> matrix = { 0 };
    NTL::ZZ modulus;

    initMatrix(argv[1], argv[2], argv[3], modulus, matrix);
    NTL::ZZ_p::init(modulus);

    uint16_t * d_GF216_log_table;//65536
    uint16_t * d_GF216_exp_table;//131070

    hipMalloc((void**)&d_GF216_log_table, sizeof(GF216_log_table));
    hipMemcpy(d_GF216_log_table, &GF216_log_table, sizeof(GF216_log_table),hipMemcpyHostToDevice);

    hipMalloc((void**)&d_GF216_exp_table, sizeof(GF216_exp_table));
    hipMemcpy(d_GF216_exp_table, &GF216_exp_table, sizeof(GF216_exp_table),	hipMemcpyHostToDevice);

    uintX * l_query, * d_query;
    hipHostMalloc((void**)&l_query, nstreams * matrix.nrows * sizeof(uintX));
    hipMalloc((void**)&d_query, nstreams * matrix.nrows * sizeof(uintX));

    uint16_t * l_response, * d_response;
    hipHostMalloc((void**)&l_response, nstreams * matrix.ncols * sizeof(uint16_t));
    hipMalloc((void**)&d_response, nstreams * matrix.ncols * sizeof(uint16_t));
    uint16_t * cpu_l_response = (uint16_t*) malloc(nstreams*matrix.ncols * sizeof(uint16_t));

    hipStream_t * streams = new hipStream_t[nstreams];
    for (int i = 0; i < nstreams; ++i) hipStreamCreate(&streams[i]);

    for (int i = 0; i < nstreams * matrix.nrows; i++)
    {
	l_query[i] = NTL::RandomBits_long(16);
    }

    std::atomic<int> cnt = ATOMIC_VAR_INIT(0);
    auto start = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds onesec{1000000000};
    //while (std::chrono::duration_cast<std::chrono::duration<int,std::nano>>(std::chrono::high_resolution_clock::now() - start) < onesec)
    {
	//int i = cnt % nstreams;
	#pragma omp parallel
	for (int i = 0; i < nstreams; i++)
	{
	    uint16_t * __l_response = l_response + i * matrix.ncols;
	    uint16_t * __d_response = d_response + i * matrix.ncols;
	    uintX * __l_query = l_query + i * matrix.nrows;
	    uintX * __d_query = d_query + i * matrix.nrows;		

	    SpMV(u_wrap,__l_response, __l_query, __d_response,
		__d_query, streams[i], matrix, d_GF216_log_table, d_GF216_exp_table);
	    SpMV_gf216_cpu(u_wrap, cpu_l_response, __l_query, matrix);


	    std::atomic_fetch_add(&cnt, 1);
	    //for (int j = 0; j < matrix.nrows; j++)
	    //{	__l_query[j] = 65535;
		//to_uint<uintX>(NTL::rep(NTL::random_ZZ_p()), __l_query[j]);
		//NTL::BytesFromZZ((unsigned char *)&__l_query[j], NTL::RandomPrime_ZZ(8), 1);//conv<uint>();
	    //}
	}
    }

    std::cout << "Count: " << cnt << "\n";

    // cleanup
    for (int i = 0; i < nstreams; ++i) hipStreamDestroy(streams[i]);
    delete [] streams;
    hipHostFree(l_query);
    hipFree(d_query);
    hipHostFree(l_response);
    hipFree(d_response);
    hipFree(d_GF216_log_table);
    hipFree(d_GF216_exp_table);
    freeMatrix<uint>(matrix);
    return 0;
}



